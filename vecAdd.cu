
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    int size = 100;  // Size of the vectors
    int* a, * b, * c;    // Host vectors
    int* dev_a, * dev_b, * dev_c;  // Device vectors

    // Allocate memory for host vectors
    a = (int*)malloc(size * sizeof(int));
    b = (int*)malloc(size * sizeof(int));
    c = (int*)malloc(size * sizeof(int));

    // Initialize host vectors
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Allocate memory on the device for device vectors
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel for vector addition
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);

    // Copy result from device to host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < size; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}